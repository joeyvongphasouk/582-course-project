#include "hip/hip_runtime.h"
#include "putil.cu"

__device__ inline void paddstate(List *, State *, List *);
__device__ inline void pstep(List *, int, List *);

__device__ __shared__ char buf[8000];

__device__ inline int pstrlen(char *str) {
  int len = 0;
  while (*str != 0) {
    len++;
    str += 1;
  }
  return len;
}

/*
 * Convert infix regexp re to postfix notation.
 * Insert ESC (or 0x1b) as explicit concatenation operator.
 * Cheesy parser, return static buffer.
 */
__device__ inline char *pre2post(char *re) {
  int nalt, natom;
  char *dst;
  struct {
    int nalt;
    int natom;
  } paren[100], *p;

  p = paren;
  dst = buf;
  nalt = 0;
  natom = 0;

  int len = pstrlen(re);
  if (len >= sizeof buf / 2)
    return NULL;
  for (; *re; re++) {
    switch (*re) {
    case PAREN_OPEN: // (
      if (natom > 1) {
        --natom;
        *dst++ = CONCATENATE;
      }
      if (p >= paren + 100)
        return NULL;
      p->nalt = nalt;
      p->natom = natom;
      p++;
      nalt = 0;
      natom = 0;
      break;
    case ALTERNATE: // |
      if (natom == 0)
        return NULL;
      while (--natom > 0)
        *dst++ = CONCATENATE;
      nalt++;
      break;
    case PAREN_CLOSE: // )
      if (p == paren)
        return NULL;
      if (natom == 0)
        return NULL;
      while (--natom > 0)
        *dst++ = CONCATENATE;
      for (; nalt > 0; nalt--)
        *dst++ = ALTERNATE;
      --p;
      nalt = p->nalt;
      natom = p->natom;
      natom++;
      break;
    case STAR:     // *
    case PLUS:     // +
    case QUESTION: // ?
      if (natom == 0)
        return NULL;
      *dst++ = *re;
      break;
    default:
      if (natom > 1) {
        --natom;
        *dst++ = CONCATENATE;
      }
      *dst++ = *re;
      natom++;
      break;
    }
  }
  if (p != paren)
    return NULL;
  while (--natom > 0)
    *dst++ = CONCATENATE;
  for (; nalt > 0; nalt--)
    *dst++ = ALTERNATE;
  *dst = 0;

  return dst;
}

/* Compute initial state list */
__device__ inline List *pstartlist(State *start, List *l) {
  l->n = 0;

  List addStartState;
  paddstate(l, start, &addStartState);
  return l;
}

/* Check whether state list contains a match. */
__device__ inline int ispmatch(List *l) {
  int i;

  for (i = 0; i < l->n; i++) {
    if (l->s[i]->c == Match)
      return 1;
  }
  return 0;
}

/* Add s to l, following unlabeled arrows. */
__device__ inline void paddstate(List *l, State *s, List *addStateList) {
  addStateList->n = 0;
  PUSH(addStateList, s);
  /* follow unlabeled arrows */
  while (!IS_EMPTY(addStateList)) {

    s = POP(addStateList);

    // lastlist check is present to ensure that if
    // multiple states point to this state, then only
    // one instance of the state is added to the list
    if (s == NULL)
      ;
    else if (s->c == Split) {
      PUSH(addStateList, s->out);
      PUSH(addStateList, s->out1);
    } else {
      l->s[l->n++] = s;
    }
  }
}

/*
 * pstep the NFA from the states in clist
 * past the character c,
 * to create next NFA state set nlist.
 */
__device__ inline void pstep(List *clist, int c, List *nlist) {
  int i;
  State *s;
  nlist->n = 0;
  for (i = 0; i < clist->n; i++) {
    s = clist->s[i];

    if (s->c == c || s->c == Any) {
      List addStartState;
      paddstate(nlist, s->out, &addStartState);
    }
  }
}

/* Run NFA to determine whether it matches s. */
__device__ inline int pmatch(State *start, char *s, List *dl1, List *dl2) {
  int c;
  List *clist, *nlist, *t;

  clist = pstartlist(start, dl1);
  nlist = dl2;
  for (; *s; s++) {
    c = *s & 0xFF;
    pstep(clist, c, nlist);
    t = clist;
    clist = nlist;
    nlist = t; // swap clist, nlist

    // check for a match in the middle of the string
    //	if (ispmatch(clist))
    // return 1;
  }
  return ispmatch(clist);
}

/* Check for a string match at all possible start positions */
__device__ inline int panypmatch(State *start, char *s, List *dl1, List *dl2) {
  int isMatch = pmatch(start, s, dl1, dl2);
  /*	int index = 0;
          int len = pstrlen(s);
          while (!isMatch && index < len) {
                  isMatch = pmatch(start, s + index, dl1, dl2);
                  index ++;
          }
  */
  return isMatch;
}

__device__ __shared__ State *st;
__device__ __shared__ State s[100];

__global__ void parallelMatch(char *bigLine, u32 *tableOfLineStarts,
                              int numLines, int numRegexs, int time,
                              char *regexLines, u32 *regexTable,
                              unsigned char *devResult) {

  for (int j = 0; j < numRegexs; j++) {
    // printf("%s\n", regexLines + regexTable[i]);

    if (threadIdx.x == 0) {
      pre2post(regexLines + regexTable[j]);

      char *postfix = buf;

      pnstate = 0;
      states = s;

      st = ppost2nfa(postfix);
    }

    __syncthreads();

    List d1;
    List d2;

    int i;
    for (i = blockIdx.x * blockDim.x + threadIdx.x; i < numLines;
         i += gridDim.x * blockDim.x) {

      char *lineSegment = bigLine + tableOfLineStarts[i];
      if (panypmatch(st, lineSegment, &d1, &d2))
        devResult[i] = 1;
      else
        devResult[i] = 0;
    }
  }
}

__global__ void test(char *bigLine, u32 *tableOfLineStarts, int numLines,
                     unsigned char *devResult) {
  for (int k = 0; k < 2000; k++) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numLines;
         i += gridDim.x * blockDim.x) {
      char *lineSegment = bigLine + tableOfLineStarts[i];
      for (int j = 0; j < pstrlen(lineSegment); j++) {
        if (lineSegment[i] == '\n')
          devResult[i] = 1;
        else
          devResult[i] = 0;
        break;
      }
    }
  }
}

void pMatch(char *bigLine, u32 *tableOfLineStarts, int numLines, int numRegexs,
            int time, char *regexLines, u32 *regexTable, char **lines,
            u32 *hostLineStarts) {

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(parallelMatch), hipFuncCachePreferShared);

  unsigned char *devResult;
  hipMalloc(&devResult, numLines * sizeof(unsigned char));

  parallelMatch<<<512, 160>>>(bigLine, tableOfLineStarts, numLines, numRegexs,
                              time, regexLines, regexTable, devResult);
  // test<<<512,160>>>(bigLine, tableOfLineStarts, numLines, devResult);
  hipDeviceSynchronize();

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  if (!time) {
    unsigned char *hostResult =
        (unsigned char *)malloc(numLines * sizeof(unsigned char));
    hipMemcpy(hostResult, devResult, numLines * sizeof(unsigned char),
               hipMemcpyDeviceToHost);

    for (int i = 0; i < numLines; i++) {
      if (hostResult[i] == 1)
        PRINT(time, "%s\n", lines[0] + hostLineStarts[i]); //[i % numLines]);
    }
  }

  hipFree(devResult);
  hipFree(bigLine);
  hipFree(tableOfLineStarts);
}
