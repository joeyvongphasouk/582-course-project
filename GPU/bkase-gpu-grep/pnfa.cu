#include "hip/hip_runtime.h"
// #include "putil.cu"

// __device__ inline void paddstate(List *, State *, List *);
// __device__ inline void pstep(List *, int, List *);


// // __device__ __shared__ char buf[8000];

// __device__ inline int pstrlen(char *str) {
//   int len = 0;
//   while (*str != 0) {
//     len++;
//     str += 1;
//   }
//   return len;
// }

// /*
//  * Convert infix regexp re to postfix notation.
//  * Insert ESC (or 0x1b) as explicit concatenation operator.
//  * Cheesy parser, return static buffer.
//  */
// __device__ inline char *pre2post(char *re, char *dst) {
//   int nalt, natom;
//   struct {
//     int nalt;
//     int natom;
//   } paren[100], *p;
//   p = paren;
//   nalt = 0;
//   natom = 0;
//   char *start = dst;

//   int len = pstrlen(re);
//   if (len >= 256)
//     return NULL;
//   for (; *re; re++) {
//     switch (*re) {
//     case PAREN_OPEN: // (
//       if (natom > 1) {
//         --natom;
//         *dst++ = CONCATENATE;
//       }
//       if (p >= paren + 100)
//         return NULL;
//       p->nalt = nalt;
//       p->natom = natom;
//       p++;
//       nalt = 0;
//       natom = 0;
//       break;
//     case ALTERNATE: // |
//       if (natom == 0)
//         return NULL;
//       while (--natom > 0)
//         *dst++ = CONCATENATE;
//       nalt++;
//       break;
//     case PAREN_CLOSE: // )
//       if (p == paren)
//         return NULL;
//       if (natom == 0)
//         return NULL;
//       while (--natom > 0)
//         *dst++ = CONCATENATE;
//       for (; nalt > 0; nalt--)
//         *dst++ = ALTERNATE;
//       --p;
//       nalt = p->nalt;
//       natom = p->natom;
//       natom++;
//       break;
//     case STAR:     // *
//     case PLUS:     // +
//     case QUESTION: // ?
//       if (natom == 0)
//         return NULL;
//       *dst++ = *re;
//       break;
//     default:
//       if (natom > 1) {
//         --natom;
//         *dst++ = CONCATENATE;
//       }
//       *dst++ = *re;
//       natom++;
//       break;
//     }
//   }
//   if (p != paren)
//     return NULL;
//   while (--natom > 0)
//     *dst++ = CONCATENATE;
//   for (; nalt > 0; nalt--)
//     *dst++ = ALTERNATE;
//   *dst = 0;

//   return start;
// }

// /* Compute initial state list */
// __device__ inline List *pstartlist(State *start, List *l) {
//   l->n = 0;

//   List addStartState;
//   paddstate(l, start, &addStartState);
//   return l;
// }

// /* Check whether state list contains a match. */
// __device__ inline int ispmatch(List *l) {
//   int i;

//   for (i = 0; i < l->n; i++) {
//     if (l->s[i]->c == Match)
//       return 1;
//   }
//   return 0;
// }

// /* Add s to l, following unlabeled arrows. */
// __device__ inline void paddstate(List *l, State *s, List *addStateList) {
//   addStateList->n = 0;
//   PUSH(addStateList, s);
//   /* follow unlabeled arrows */
//   while (!IS_EMPTY(addStateList)) {

//     s = POP(addStateList);

//     // lastlist check is present to ensure that if
//     // multiple states point to this state, then only
//     // one instance of the state is added to the list
//     if (s == NULL)
//       ;
//     else if (s->c == Split) {
//       PUSH(addStateList, s->out);
//       PUSH(addStateList, s->out1);
//     } else {
//       l->s[l->n++] = s;
//     }
//   }
// }

// /*
//  * pstep the NFA from the states in clist
//  * past the character c,
//  * to create next NFA state set nlist.
//  */
// __device__ inline void pstep(List *clist, int c, List *nlist) {
//   int i;
//   State *s;
//   nlist->n = 0;
//   for (i = 0; i < clist->n; i++) {
//     s = clist->s[i];

//     if (s->c == c || s->c == Any) {
//       List addStartState;
//       paddstate(nlist, s->out, &addStartState);
//     }
//   }
// }

// /* Run NFA to determine whether it matches s. */
// __device__ inline int pmatch(State *start, char *s, List *dl1, List *dl2) {
//   int c;
//   List *clist, *nlist, *t;

//   clist = pstartlist(start, dl1);
//   nlist = dl2;
//   for (; *s; s++) {
//     c = *s & 0xFF;
//     pstep(clist, c, nlist);
//     t = clist;
//     clist = nlist;
//     nlist = t; // swap clist, nlist

//     // check for a match in the middle of the string
//     //	if (ispmatch(clist))
//     // return 1;
//   }
//   return ispmatch(clist);
// }

// /* Check for a string match at all possible start positions */
// __device__ inline int panypmatch(State *start, char *s, List *dl1, List *dl2) {
//   int isMatch = pmatch(start, s, dl1, dl2);
//   /*	int index = 0;
//           int len = pstrlen(s);
//           while (!isMatch && index < len) {
//                   isMatch = pmatch(start, s + index, dl1, dl2);
//                   index ++;
//           }
//   */
//   return isMatch;
// }
// __device__ State **regexNFAs;

// __global__ void parallelMatch(char *bigLine, u32 *tableOfLineStarts,
//                               int numLines, int numRegexs, int time,
//                               char *regexLines, u32 *regexTable,
//                               unsigned char *devResult) {

//   extern __shared__ char sharedMem[];
//   State **st_shared = (State **)sharedMem;
//   char *postfixBuf = (char *)(st_shared + 1);

//   for (int j = 0; j < numRegexs; j++) {
//       if (threadIdx.x == 0) {
//           char *re = regexLines + regexTable[j];
//           char *postfix = pre2post(re, postfixBuf);

    
//       pnstate = 0;
//       states = regexNFAs[j];  // assign per-regex memory
    
//       State *localSt = ppost2nfa(postfix);
    
//       // Use shared memory pointer to pass start state to other threads
//       st_shared[0] = localSt;
//     }
//     __syncthreads();
    
//     State *st = st_shared[0];
    
//     __syncthreads();

//     List d1;
//     List d2;

//     int i;
//     for (i = blockIdx.x * blockDim.x + threadIdx.x; i < numLines;
//          i += gridDim.x * blockDim.x) {
      
//       // modified so that the devResult would pass if already matched
//       // ensures ruleset matches over everything
//       if (devResult[i] == 1) continue;
//       char *lineSegment = bigLine + tableOfLineStarts[i];
//       if (panypmatch(st, lineSegment, &d1, &d2))
//         devResult[i] = 1;
//     }
//   }
// }

// __global__ void test(char *bigLine, u32 *tableOfLineStarts, int numLines,
//                      unsigned char *devResult) {
//   for (int k = 0; k < 2000; k++) {
//     for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numLines;
//          i += gridDim.x * blockDim.x) {
//       char *lineSegment = bigLine + tableOfLineStarts[i];
//       for (int j = 0; j < pstrlen(lineSegment); j++) {
//         if (lineSegment[i] == '\n')
//           devResult[i] = 1;
//         else
//           devResult[i] = 0;
//         break;
//       }
//     }
//   }
// }

// void pMatch(char *bigLine, u32 *tableOfLineStarts, int numLines, int numRegexs,
//             int time, char *regexLines, u32 *regexTable, char **lines,
//             u32 *hostLineStarts) {

//   hipFuncSetCacheConfig(reinterpret_cast<const void*>(parallelMatch), hipFuncCachePreferShared);


//   // new changes -----------
//   int maxStatesPerRegex = 100; // arbitrary max state count

//   // alloc array for nfa pointers on host and device
//   State **h_regexNFAs = (State **)malloc(numRegexs * sizeof(State *));
//   for (int i = 0; i < numRegexs; i++) {
//     hipMalloc((void **)&h_regexNFAs[i], maxStatesPerRegex * sizeof(State));
//   }
//   State **d_regexNFAs;
//   hipMalloc(&d_regexNFAs, numRegexs * sizeof(State *));

//   // copy over pointers
//   hipMemcpy(d_regexNFAs, h_regexNFAs, numRegexs * sizeof(State *), hipMemcpyHostToDevice);
//   hipMemcpyToSymbol(HIP_SYMBOL(regexNFAs), &d_regexNFAs, sizeof(State **));
//   // changes end -----------


//   unsigned char *devResult;
//   hipMalloc(&devResult, numLines * sizeof(unsigned char));
//   size_t sharedMemSize = sizeof(State *) + 256 * sizeof(char);
//   parallelMatch<<<512, 160, sharedMemSize>>>(bigLine, tableOfLineStarts, numLines, numRegexs,
//                               time, regexLines, regexTable, devResult);
//   // parallelMatch<<<512, 160>>>(bigLine, tableOfLineStarts, numLines, numRegexs,
//   //   time, regexLines, regexTable, devResult);
//   // test<<<512,160>>>(bigLine, tableOfLineStarts, numLines, devResult);
//   hipDeviceSynchronize();

//   hipError_t error = hipGetLastError();
//   if (error != hipSuccess) {
//     printf("CUDA Error: %s\n", hipGetErrorString(error));
//     exit(-1);
//   }

//   if (!time) {
//     unsigned char *hostResult =
//         (unsigned char *)malloc(numLines * sizeof(unsigned char));
//     hipMemcpy(hostResult, devResult, numLines * sizeof(unsigned char),
//                hipMemcpyDeviceToHost);

//     for (int i = 0; i < numLines; i++) {
//       if (hostResult[i] == 1)
//         PRINT(time, "%s\n", lines[0] + hostLineStarts[i]); //[i % numLines]);
//     }
//   }

//   hipFree(devResult);
//   hipFree(bigLine);
//   hipFree(tableOfLineStarts);

//   // free newly added states as well
//   for (int i = 0; i < numRegexs; i++) {
//     hipFree(h_regexNFAs[i]);
//   }
//   free(h_regexNFAs);
//   hipFree(d_regexNFAs);
// }


#include "putil.cu"

__device__ inline void paddstate(List *, State *, List *);
__device__ inline void pstep(List *, int, List *);

__device__ __shared__ char buf[8000];

__device__ inline int pstrlen(char *str) {
  int len = 0;
  while (*str != 0) {
    len++;
    str += 1;
  }
  return len;
}

/*
 * Convert infix regexp re to postfix notation.
 * Insert ESC (or 0x1b) as explicit concatenation operator.
 * Cheesy parser, return static buffer.
 */
__device__ inline char *pre2post(char *re) {
  int nalt, natom;
  char *dst;
  struct {
    int nalt;
    int natom;
  } paren[100], *p;

  p = paren;
  dst = buf;
  nalt = 0;
  natom = 0;

  int len = pstrlen(re);
  if (len >= sizeof buf / 2)
    return NULL;
  for (; *re; re++) {
    switch (*re) {
    case PAREN_OPEN: // (
      if (natom > 1) {
        --natom;
        *dst++ = CONCATENATE;
      }
      if (p >= paren + 100)
        return NULL;
      p->nalt = nalt;
      p->natom = natom;
      p++;
      nalt = 0;
      natom = 0;
      break;
    case ALTERNATE: // |
      if (natom == 0)
        return NULL;
      while (--natom > 0)
        *dst++ = CONCATENATE;
      nalt++;
      break;
    case PAREN_CLOSE: // )
      if (p == paren)
        return NULL;
      if (natom == 0)
        return NULL;
      while (--natom > 0)
        *dst++ = CONCATENATE;
      for (; nalt > 0; nalt--)
        *dst++ = ALTERNATE;
      --p;
      nalt = p->nalt;
      natom = p->natom;
      natom++;
      break;
    case STAR:     // *
    case PLUS:     // +
    case QUESTION: // ?
      if (natom == 0)
        return NULL;
      *dst++ = *re;
      break;
    default:
      if (natom > 1) {
        --natom;
        *dst++ = CONCATENATE;
      }
      *dst++ = *re;
      natom++;
      break;
    }
  }
  if (p != paren)
    return NULL;
  while (--natom > 0)
    *dst++ = CONCATENATE;
  for (; nalt > 0; nalt--)
    *dst++ = ALTERNATE;
  *dst = 0;

  return dst;
}

/* Compute initial state list */
__device__ inline List *pstartlist(State *start, List *l) {
  l->n = 0;

  List addStartState;
  paddstate(l, start, &addStartState);
  return l;
}

/* Check whether state list contains a match. */
__device__ inline int ispmatch(List *l) {
  int i;

  for (i = 0; i < l->n; i++) {
    if (l->s[i]->c == Match)
      return 1;
  }
  return 0;
}

/* Add s to l, following unlabeled arrows. */
__device__ inline void paddstate(List *l, State *s, List *addStateList) {
  addStateList->n = 0;
  PUSH(addStateList, s);
  /* follow unlabeled arrows */
  while (!IS_EMPTY(addStateList)) {

    s = POP(addStateList);

    // lastlist check is present to ensure that if
    // multiple states point to this state, then only
    // one instance of the state is added to the list
    if (s == NULL)
      ;
    else if (s->c == Split) {
      PUSH(addStateList, s->out);
      PUSH(addStateList, s->out1);
    } else {
      l->s[l->n++] = s;
    }
  }
}

/*
 * pstep the NFA from the states in clist
 * past the character c,
 * to create next NFA state set nlist.
 */
__device__ inline void pstep(List *clist, int c, List *nlist) {
  int i;
  State *s;
  nlist->n = 0;
  for (i = 0; i < clist->n; i++) {
    s = clist->s[i];

    if (s->c == c || s->c == Any) {
      List addStartState;
      paddstate(nlist, s->out, &addStartState);
    }
  }
}

/* Run NFA to determine whether it matches s. */
__device__ inline int pmatch(State *start, char *s, List *dl1, List *dl2) {
  int c;
  List *clist, *nlist, *t;

  clist = pstartlist(start, dl1);
  nlist = dl2;
  for (; *s; s++) {
    c = *s & 0xFF;
    pstep(clist, c, nlist);
    t = clist;
    clist = nlist;
    nlist = t; // swap clist, nlist

    // check for a match in the middle of the string
    //	if (ispmatch(clist))
    // return 1;
  }
  return ispmatch(clist);
}

/* Check for a string match at all possible start positions */
__device__ inline int panypmatch(State *start, char *s, List *dl1, List *dl2) {
  int isMatch = pmatch(start, s, dl1, dl2);
  /*	int index = 0;
          int len = pstrlen(s);
          while (!isMatch && index < len) {
                  isMatch = pmatch(start, s + index, dl1, dl2);
                  index ++;
          }
  */
  return isMatch;
}

__device__ __shared__ State *st;
__device__ __shared__ State s[100];

__global__ void parallelMatch(char *bigLine, u32 *tableOfLineStarts,
                              int numLines, int numRegexs, int time,
                              char *regexLines, u32 *regexTable,
                              unsigned char *devResult) {

  for (int j = 0; j < numRegexs; j++) {
    // printf("%s\n", regexLines + regexTable[i]);

    if (threadIdx.x == 0) {
      pre2post(regexLines + regexTable[j]);

      char *postfix = buf;

      pnstate = 0;
      states = s;

      st = ppost2nfa(postfix);
    }

    __syncthreads();

    List d1;
    List d2;

    int i;
    for (i = blockIdx.x * blockDim.x + threadIdx.x; i < numLines;
         i += gridDim.x * blockDim.x) {
      
      // modified so that the devResult would pass if already matched
      // ensures ruleset matches over everything
      if (devResult[i] == 1) continue;
      char *lineSegment = bigLine + tableOfLineStarts[i];
      if (panypmatch(st, lineSegment, &d1, &d2))
        devResult[i] = 1;
    }
    
    __syncthreads();
  }
}

__global__ void test(char *bigLine, u32 *tableOfLineStarts, int numLines,
                     unsigned char *devResult) {
  for (int k = 0; k < 2000; k++) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numLines;
         i += gridDim.x * blockDim.x) {
      char *lineSegment = bigLine + tableOfLineStarts[i];
      for (int j = 0; j < pstrlen(lineSegment); j++) {
        if (lineSegment[i] == '\n')
          devResult[i] = 1;
        else
          devResult[i] = 0;
        break;
      }
    }
  }
}

void pMatch(char *bigLine, u32 *tableOfLineStarts, int numLines, int numRegexs,
            int time, char *regexLines, u32 *regexTable, char **lines,
            u32 *hostLineStarts) {

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(parallelMatch), hipFuncCachePreferShared);

  unsigned char *devResult;
  hipMalloc(&devResult, numLines * sizeof(unsigned char));

  parallelMatch<<<512, 160>>>(bigLine, tableOfLineStarts, numLines, numRegexs,
                              time, regexLines, regexTable, devResult);
  // test<<<512,160>>>(bigLine, tableOfLineStarts, numLines, devResult);
  hipDeviceSynchronize();

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  if (!time) {
    unsigned char *hostResult =
        (unsigned char *)malloc(numLines * sizeof(unsigned char));
    hipMemcpy(hostResult, devResult, numLines * sizeof(unsigned char),
               hipMemcpyDeviceToHost);

    for (int i = 0; i < numLines; i++) {
      if (hostResult[i] == 1)
        PRINT(time, "%s\n", lines[0] + hostLineStarts[i]); //[i % numLines]);
    }
  }

  hipFree(devResult);
  hipFree(bigLine);
  hipFree(tableOfLineStarts);
}