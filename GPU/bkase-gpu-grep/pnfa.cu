#include "hip/hip_runtime.h"
#include "putil.cu"
#include <vector>
#include <set>

__device__ inline void paddstate(List *, State *, List *);
__device__ inline void pstep(List *, int, List *);

__device__ __shared__ char buf[16384];

__device__ inline int pstrlen(char *str) {
  int len = 0;
  while (*str != 0) {
    len++;
    str += 1;
  }
  return len;
}

/*
 * Convert infix regexp re to postfix notation.
 * Insert ESC (or 0x1b) as explicit concatenation operator.
 * Cheesy parser, return static buffer.
 */
__device__ inline char *pre2post(char *re) {
  int nalt = 0, natom = 0;
  char *dst = buf;
  struct {
    int nalt;
    int natom;
  } paren[128], *p = paren;

  int len = pstrlen(re);
  if (len >= sizeof(buf) / 2) return NULL;

  for (; *re; re++) {
    switch (*re) {
    case PAREN_OPEN: // (
      if (natom > 1) {
        --natom;
        *dst++ = CONCATENATE;
      }
      if (p >= paren + 128) return NULL;
      p->nalt = nalt;
      p->natom = natom;
      p++;
      nalt = 0;
      natom = 0;
      break;
    case ALTERNATE: // |
      if (natom == 0) return NULL;
      while (--natom > 0) *dst++ = CONCATENATE;
      nalt++;
      break;
    case PAREN_CLOSE: // )
      if (p == paren) return NULL;
      if (natom == 0) return NULL;
      while (--natom > 0) *dst++ = CONCATENATE;
      for (; nalt > 0; nalt--) *dst++ = ALTERNATE;
      --p;
      nalt = p->nalt;
      natom = p->natom;
      natom++;
      break;
    case STAR:     // *
    case PLUS:     // +
    case QUESTION: // ?
      if (natom == 0) return NULL;
      *dst++ = *re;
      break;
    default:
      if (natom > 1) {
        --natom;
        *dst++ = CONCATENATE;
      }
      *dst++ = *re;
      natom++;
      break;
    }
  }
  if (p != paren) return NULL;
  while (--natom > 0) *dst++ = CONCATENATE;
  for (; nalt > 0; nalt--) *dst++ = ALTERNATE;
  *dst = 0;

  return dst;
}

/* Compute initial state list */
__device__ inline List *pstartlist(State *start, List *l) {
  l->n = 0;

  List addStartState;
  paddstate(l, start, &addStartState);
  return l;
}

/* Check whether state list contains a match. */
__device__ inline int ispmatch(List *l) {
  int i;

  for (i = 0; i < l->n; i++) {
    if (l->s[i]->c == Match)
      return 1;
  }
  return 0;
}

/* Add s to l, following unlabeled arrows. */
__device__ inline void paddstate(List *l, State *s, List *addStateList) {
  addStateList->n = 0;
  PUSH(addStateList, s);
  /* follow unlabeled arrows */
  while (!IS_EMPTY(addStateList)) {

    s = POP(addStateList);

    // lastlist check is present to ensure that if
    // multiple states point to this state, then only
    // one instance of the state is added to the list
    if (s == NULL)
      ;
    else if (s->c == Split) {
      PUSH(addStateList, s->out);
      PUSH(addStateList, s->out1);
    } else {
      l->s[l->n++] = s;
    }
  }
}

/*
 * pstep the NFA from the states in clist
 * past the character c,
 * to create next NFA state set nlist.
 */
__device__ inline void pstep(List *clist, int c, List *nlist) {
  int i;
  State *s;
  nlist->n = 0;
  for (i = 0; i < clist->n; i++) {
    s = clist->s[i];

    if (s->c == c || s->c == Any) {
      List addStartState;
      paddstate(nlist, s->out, &addStartState);
    }
  }
}

/* Run NFA to determine whether it matches s. */
__device__ inline int pmatch(State *start, char *s, List *dl1, List *dl2) {
    int c;
    List *clist, *nlist, *t;

    clist = pstartlist(start, dl1);
    nlist = dl2;
    
    // Pre-compute string length
    int len = 0;
    while (s[len] != '\0') len++;
    
    // Early exit for empty strings
    if (len == 0) return 0;
    
    for (int i = 0; i < len; i++) {
        c = s[i] & 0xFF;
        pstep(clist, c, nlist);
        t = clist;
        clist = nlist;
        nlist = t;

        // Early exit if we find a match
        if (ispmatch(clist))
            return 1;
    }
    return 0;
}

/* Check for a string match at all possible start positions */
__device__ inline int panypmatch(State *start, char *s, List *dl1, List *dl2) {
    // Pre-compute string length
    int len = 0;
    while (s[len] != '\0') len++;
    
    // Early exit for empty strings
    if (len == 0) return 0;
    
    // Try matching at each position
    for (int i = 0; i < len; i++) {
        if (pmatch(start, s + i, dl1, dl2)) {
            return 1;
        }
    }
    return 0;
}

__device__ __shared__ State *st;
__device__ __shared__ State s[128];

// Add host version of pre2post
char* host_pre2post(char* re) {
    static char buf[16384];
    int nalt = 0, natom = 0;
    char *dst = buf;
    struct {
        int nalt;
        int natom;
    } paren[128], *p = paren;

    int len = strlen(re);
    if (len >= sizeof(buf) / 2) return NULL;

    for (; *re; re++) {
        switch (*re) {
        case PAREN_OPEN: // (
            if (natom > 1) {
                --natom;
                *dst++ = CONCATENATE;
            }
            if (p >= paren + 128) return NULL;
            p->nalt = nalt;
            p->natom = natom;
            p++;
            nalt = 0;
            natom = 0;
            break;
        case ALTERNATE: // |
            if (natom == 0) return NULL;
            while (--natom > 0) *dst++ = CONCATENATE;
            nalt++;
            break;
        case PAREN_CLOSE: // )
            if (p == paren) return NULL;
            if (natom == 0) return NULL;
            while (--natom > 0) *dst++ = CONCATENATE;
            for (; nalt > 0; nalt--) *dst++ = ALTERNATE;
            --p;
            nalt = p->nalt;
            natom = p->natom;
            natom++;
            break;
        case STAR:     // *
        case PLUS:     // +
        case QUESTION: // ?
            if (natom == 0) return NULL;
            *dst++ = *re;
            break;
        default:
            if (natom > 1) {
                --natom;
                *dst++ = CONCATENATE;
            }
            *dst++ = *re;
            natom++;
            break;
        }
    }
    if (p != paren) return NULL;
    while (--natom > 0) *dst++ = CONCATENATE;
    for (; nalt > 0; nalt--) *dst++ = ALTERNATE;
    *dst = 0;

    return buf;
}

// Add new host function for preprocessing
void preprocessRegex(char* regex, State** devStates, int* devNState) {
    if (!regex || !devStates || !devNState) {
        return;
    }
    
    // Convert regex to postfix on host
    char* postfix = host_pre2post(regex);
    if (!postfix) {
        return;
    }
    
    // Convert postfix to NFA on host
    State* start = post2nfa(postfix);
    if (!start) {
        return;
    }
    
    // Initialize the dev pointers in the NFA
    // First pass: count states and initialize dev pointers
    int stateCount = 0;
    State* current = start;
    std::set<State*> visited;
    
    // Use a stack for DFS traversal
    std::vector<State*> stack;
    stack.push_back(start);
    
    while (!stack.empty()) {
        current = stack.back();
        stack.pop_back();
        
        if (visited.find(current) != visited.end()) {
            continue;
        }
        
        visited.insert(current);
        current->dev = NULL;
        stateCount++;
        
        if (current->c == Split) {
            if (current->out != NULL) {
                stack.push_back(current->out);
            }
            if (current->out1 != NULL) {
                stack.push_back(current->out1);
            }
        } else if (current->out != NULL) {
            stack.push_back(current->out);
        }
    }
    
    // Allocate device memory for all states
    State* dev_states;
    hipError_t err = hipMalloc(&dev_states, stateCount * sizeof(State));
    if (err != hipSuccess) {
        return;
    }
    
    // Second pass: copy states to device and set up pointers
    visited.clear();
    stack.clear();
    stack.push_back(start);
    int stateIndex = 0;
    
    while (!stack.empty()) {
        current = stack.back();
        stack.pop_back();
        
        if (visited.find(current) != visited.end()) {
            continue;
        }
        
        visited.insert(current);
        
        // Copy state to device
        err = hipMemcpy(dev_states + stateIndex, current, sizeof(State), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            return;
        }
        current->dev = dev_states + stateIndex;
        stateIndex++;
        
        if (current->c == Split) {
            if (current->out != NULL) {
                stack.push_back(current->out);
            }
            if (current->out1 != NULL) {
                stack.push_back(current->out1);
            }
        } else if (current->out != NULL) {
            stack.push_back(current->out);
        }
    }
    
    // Third pass: update pointers in device memory
    visited.clear();
    stack.clear();
    stack.push_back(start);
    
    while (!stack.empty()) {
        current = stack.back();
        stack.pop_back();
        
        if (visited.find(current) != visited.end()) {
            continue;
        }
        
        visited.insert(current);
        
        if (current->dev != NULL) {
            if (current->out != NULL) {
                err = hipMemcpy(&(current->dev->out), &(current->out->dev), sizeof(State*), hipMemcpyHostToDevice);
                if (err != hipSuccess) {
                    return;
                }
            }
            if (current->out1 != NULL) {
                err = hipMemcpy(&(current->dev->out1), &(current->out1->dev), sizeof(State*), hipMemcpyHostToDevice);
                if (err != hipSuccess) {
                    return;
                }
            }
            
            if (current->c == Split) {
                if (current->out != NULL) {
                    stack.push_back(current->out);
                }
                if (current->out1 != NULL) {
                    stack.push_back(current->out1);
                }
            } else if (current->out != NULL) {
                stack.push_back(current->out);
            }
        }
    }
    
    *devStates = dev_states;
    *devNState = stateCount;
}

__global__ void parallelMatch(char *bigLine, u32 *tableOfLineStarts,
                              int numLines, State* devStates,
                              unsigned char *devResult) {
    // Each thread gets its own state lists
    List d1, d2;
    
    // Get the start state from device memory
    State* start = devStates;
    
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numLines; 
         i += gridDim.x * blockDim.x) {
        char *lineSegment = bigLine + tableOfLineStarts[i];
        devResult[i] = panypmatch(start, lineSegment, &d1, &d2) ? 1 : 0;
    }
}

void pMatch(char *bigLine, u32 *tableOfLineStarts, int numLines, int numRegexs,
            int time, char *regexLines, u32 *regexTable, char **lines,
            u32 *hostLineStarts) {
    if (!bigLine || !tableOfLineStarts || !regexLines || !regexTable || !lines || !hostLineStarts) {
        return;
    }
    
    if (numLines <= 0 || numRegexs <= 0) {
        return;
    }
    
    // Create CUDA streams for concurrent operations
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    
    // Copy regex table value from device to host
    u32 host_regex_offset;
    hipError_t err = hipMemcpyAsync(&host_regex_offset, regexTable, sizeof(u32), 
                                    hipMemcpyDeviceToHost, stream1);
    if (err != hipSuccess) {
        return;
    }
    
    // Allocate device memory for results
    unsigned char *devResult;
    err = hipMallocAsync(&devResult, numLines * sizeof(unsigned char), stream1);
    if (err != hipSuccess) {
        return;
    }
    
    // Initialize results to 0
    err = hipMemsetAsync(devResult, 0, numLines * sizeof(unsigned char), stream1);
    if (err != hipSuccess) {
        return;
    }
    
    // Copy the regex string from device to host
    char host_regex[1024] = {0};
    int i = 0;
    char current_char;
    do {
        err = hipMemcpyAsync(&current_char, regexLines + host_regex_offset + i, 
                            sizeof(char), hipMemcpyDeviceToHost, stream2);
        if (err != hipSuccess) {
            return;
        }
        host_regex[i] = current_char;
        i++;
    } while (current_char != '\0' && i < 1023);
    
    // Wait for regex string copy to complete
    hipStreamSynchronize(stream2);
    
    // Preprocess regex and get NFA states
    State* devStates = NULL;
    int devNState = 0;
    preprocessRegex(host_regex, &devStates, &devNState);
    
    // Wait for memory operations to complete
    hipStreamSynchronize(stream1);
    
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(parallelMatch), hipFuncCachePreferL1);

    // Use maximum threads per block for better occupancy
    int blockSize = 256;
    int gridSize = (numLines + blockSize - 1) / blockSize;
    
    // Launch kernel with preprocessed states
    parallelMatch<<<gridSize, blockSize, 0, stream1>>>(bigLine, tableOfLineStarts, numLines, 
                                                     devStates, devResult);
    
    if (!time) {
        // Allocate host memory for results
        unsigned char *hostResult = (unsigned char *)malloc(numLines * sizeof(unsigned char));
        
        // Copy results back to host
        err = hipMemcpyAsync(hostResult, devResult, numLines * sizeof(unsigned char),
                           hipMemcpyDeviceToHost, stream1);
        if (err != hipSuccess) {
            free(hostResult);
            return;
        }
        
        // Wait for kernel and memory copy to complete
        hipStreamSynchronize(stream1);
        
        // Process results
        for (int i = 0; i < numLines; i++) {
            if (hostResult[i] == 1)
                printf("%s\n", lines[0] + hostLineStarts[i]);
        }
        free(hostResult);
    } else {
        // Just wait for kernel to complete
        hipStreamSynchronize(stream1);
    }

    // Cleanup
    hipFreeAsync(devStates, stream1);
    hipFreeAsync(devResult, stream1);
    
    // Wait for cleanup to complete
    hipStreamSynchronize(stream1);
    
    // Destroy streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
}

__global__ void test(char *bigLine, u32 *tableOfLineStarts, int numLines,
                     unsigned char *devResult) {
  for (int k = 0; k < 2000; k++) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numLines;
         i += gridDim.x * blockDim.x) {
      char *lineSegment = bigLine + tableOfLineStarts[i];
      for (int j = 0; j < pstrlen(lineSegment); j++) {
        if (lineSegment[i] == '\n')
          devResult[i] = 1;
        else
          devResult[i] = 0;
        break;
      }
    }
  }
}